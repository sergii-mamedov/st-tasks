#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

const double pi = 3.14159265358979323846;

void X_init(double *a, int numTicks, double delta) {
	for (int i = 0; i <= numTicks; ++i)
	for (int j = 0; j <= numTicks; ++j)
		a[i*(numTicks+1) + j] = (double)j * delta;
}

void Y_init(double *a, int numTicks, double delta) {
	for (int i = 0; i <= numTicks; ++i)
	for (int j = 0; j <= numTicks; ++j)
		a[i*(numTicks+1) + j] = (double)i * delta;
}

void array_show(double *a, int numTicks) {
	for (int i = 0; i <= numTicks; ++i) {
	for (int j = 0; j <= numTicks; ++j)
		printf("%3d ", (int)a[i*(numTicks+1) + j]);

	printf("\n");
	}
}

__global__ void kernel(double *x, double *y, double *z, double mux, double muy, double b4, int size)
{

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= size)
    	return;

    // initial values
    double x0 = (double)x[tid];
    double y0 = (double)y[tid];
    double px0 = 0.0, py0 = 0.0;

    // проверка вхождение в круг, до первой итерации
    if (x0*x0 + y0*y0 >= 1.0) {
    	z[tid] = 0.0;
    	return;
    }

    int n;
    double x1, px1, px2, y1, py1, py2;
    for (n = 0; n <= 100001; n++) {
        if (x0*x0 + y0*y0 < 1.0) {
            x1  =  x0 * cos(2.0*pi*mux) + px0 * sin(2.0*pi*mux);
            px1 = -x0 * sin(2.0*pi*mux) + px0 * cos(2.0*pi*mux);
            y1  =  y0 * cos(2.0*pi*muy) + py0 * sin(2.0*pi*muy);
            py1 = -y0 * sin(2.0*pi*muy) + py0 * cos(2.0*pi*muy);
            px2 = px1 + b4 * (x1*x1*x1 - 3.0*x1*y1*y1);
            py2 = py1 - b4 * (y1*y1*y1 - 3.0*x1*x1*y1);
            x0  =  x1;  y0 =  y1;
            px0 = px2; py0 = py2;
        } else {
        	break;
        }
    }

    n--;
    z[tid] = (double)(n);
}

int main(void)
{
	const double mux = 0.32;
	const double muy = 0.32;
	const double b4  = 0.50;
	int numTicks = 10;
	double delta = 1.0 / numTicks;
	int arraySize =(numTicks + 1) * (numTicks + 1);
	int numBytes = arraySize * sizeof(double);
	double *x, *y, *z, *x_dev, *y_dev, *z_dev;

	// allocate host memory
	x = (double *) malloc(numBytes);
	y = (double *) malloc(numBytes);
	z = (double *) malloc(numBytes);

	// allocate X, Y array
	X_init(x, numTicks, delta);
	Y_init(y, numTicks, delta);

	// allocate device memory
	hipMalloc( (void**) &x_dev, numBytes );
	hipMalloc( (void**) &y_dev, numBytes );
	hipMalloc( (void**) &z_dev, numBytes );

	// copy X, Y from host to device
	hipMemcpy( x_dev, x, numBytes, hipMemcpyHostToDevice);
	hipMemcpy( y_dev, y, numBytes, hipMemcpyHostToDevice);
	hipMemcpy( z_dev, z, numBytes, hipMemcpyHostToDevice);

	// GPU kernel
	int threadNum = 512;
	kernel <<< arraySize/threadNum + 1, threadNum >>> (x_dev, y_dev, z_dev, mux, muy, b4, arraySize);

	// copy Z from device to host
	hipMemcpy( z, z_dev, numBytes, hipMemcpyDeviceToHost);

	// show result
	array_show(z, numTicks);

	// memory free
	hipFree(x_dev); free(x);
	hipFree(y_dev); free(y);
	hipFree(z_dev); free(z);

	return 0;
}